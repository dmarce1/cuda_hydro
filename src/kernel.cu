#include "hip/hip_runtime.h"
#include <limits>
#include <future>
#include "kernel.hpp"

#ifdef _DOUBLE
#pragma message( "Compiling with double precision.")
#if  _DU_DOUBLE
using du_real = double;
#else
using du_real = float;
#pragma message( "Compiling with single precision accumulator.")
#endif
#else
#pragma message( "Compiling with single precision.")
using du_real = float;
#endif

#define NRK 2
#define CFL real(0.4)
#define DE_SWITCH_1 real(0.001)
#define DE_SWITCH_2 real(0.01)
#define THETA real(1.3)

#define INVOKE2( func, blocks, threads, ...) \
	func <<< blocks, threads >>> ( __VA_ARGS__ ); \
	hipStreamSynchronize(0)

#define INVOKE3( func, blocks, threads, size, ...) \
	func <<< blocks, threads, size >>> ( __VA_ARGS__ ); \
	hipStreamSynchronize(0)

#ifdef CARTESIAN
__device__
inline real h1(real x, real y, real z) {
	return real(1);
}
__device__
inline real h2(real x, real y, real z) {
	return real(1);
}
__device__
inline real h3(real x, real y, real z) {
	return real(1);
}
__device__
inline real hhh(real x, real y, real z) {
	return real(1);
}
#endif

#ifdef CYLINDRICAL
__device__
inline real h1(real x, real y, real z) {
	return real(1);
}
__device__
inline real h2(real x, real y, real z) {
	return x;
}
__device__
inline real h3(real x, real y, real z) {
	return real(1);
}
__device__
inline real hhh(real x, real y, real z) {
	return x;
}
#endif

#ifdef SPHERICAL
__device__
inline real h1(real x, real y, real z) {
	return real(1);
}
__device__
inline real h2(real x, real y, real z) {
	return x;
}
__device__
inline real h3(real x, real y, real z) {
	return real(1);
}
__device__
inline real hhh(real x, real y, real z) {
	return x;
}
#endif

__device__ static double* devXf[NDIM];
__device__ static double* devXc[NDIM];
static double* hostXf[NDIM];
static double* hostXc[NDIM];

__global__
void set_device_ptrs(double* xc, double* yc, double* zc, double* xf, double* yf,
		double* zf) {
	devXf[XDIM] = xf;
	devXf[YDIM] = yf;
	devXf[ZDIM] = zf;
	devXc[XDIM] = xc;
	devXc[YDIM] = yc;
	devXc[ZDIM] = zc;
}

void create_coordinates(real dx, real dy, real dz, int nx, int ny, int nz) {
	double* dXc[NDIM];
	double* dXf[NDIM];
	const int sz = nx * ny * nz;
	for (int d = 0; d != NDIM; ++d) {
		hostXf[d] = new double[sz];
		hostXc[d] = new double[sz];
		hipMalloc(&(dXf[d]), sz * sizeof(real));
		hipMalloc(&(dXc[d]), sz * sizeof(real));
	}
	real x0, y0, z0;
#ifdef CARTESIAN
	x0 = y0 = z0 = 0.0;
#endif
#ifdef CYLINDRICAL
	x0 = y0 = 0.0;
	z0 = (nz - 2 * BW + 1) * dz / 2.0;
#endif
#ifdef SPHERICAL
	x0 = y0 = 0.0;
	z0 = -1.0;
#endif
	for (int i = 0; i != nx; ++i) {
		for (int j = 0; j != ny; ++j) {
			for (int k = 0; k != nz; ++k) {
				const int iii = i + nx * (j + ny * k);
				hostXf[XDIM][iii] = (i - BW) * dx + x0;
				hostXf[YDIM][iii] = (j - BW) * dy + y0;
				hostXf[ZDIM][iii] = (k - BW) * dz + z0;
				hostXc[XDIM][iii] = (i - BW + 0.5) * dx + x0;
				hostXc[YDIM][iii] = (j - BW + 0.5) * dy + y0;
				hostXc[ZDIM][iii] = (k - BW + 0.5) * dz + z0;
			}
		}
	}
	for (int d = 0; d != NDIM; ++d) {
		hipMemcpy(&(dXf[d]), hostXf[d], sz * sizeof(real),
				hipMemcpyHostToDevice);
		hipMemcpy(&(dXc[d]), hostXc[d], sz * sizeof(real),
				hipMemcpyHostToDevice);
	}
	INVOKE2(set_device_ptrs, 1, 1,
			(dXc[XDIM]), (dXc[YDIM]), (dXc[ZDIM]), (dXf[XDIM]), (dXf[YDIM]), (dXf[ZDIM]));
}

__device__
real h(int dim, real x, real y, real z) {
	typedef real (*func_type)(real, real, real);
	static const func_type func[NDIM] = { h1, h2, h3 };
	return func[dim](x, y, z);
}

__device__
inline real minmod(real a, real b) {
	return (copysign(0.5, a) + copysign(0.5, b)) * fmin(fabs(a), fabs(b));
}

__device__
inline real minmod_theta(real a, real b) {
	return minmod(real(THETA) * minmod(a, b), real(0.5) * (a + b));
}

void cuda_exit() {
	hipDeviceReset();
}

__global__
void cuda_prep(real* U_base, real* U0_base, du_real* dU_base) {
	const int nx = blockDim.x;
	const int ny = gridDim.x;
	const int nz = gridDim.y;
	const int xi = threadIdx.x;
	const int yi = blockIdx.x;
	const int zi = blockIdx.y;
	const int nx1 = nx + 2 * BW;
	const int ny1 = ny + 2 * BW;
	const int nz1 = nz + 2 * BW;
	const int sz = nx1 * ny1 * nz1;
	const int idx = (xi + BW) + nx1 * (yi + BW) + (nx1 * ny1) * (zi + BW);
	real * U[NF];
	real * U0[NF];
	du_real* dU[NF];
	for (int f = 0; f != NF; ++f) {
		U[f] = U_base + f * sz;
		U0[f] = U0_base + f * sz;
		dU[f] = dU_base + f * sz;
	}
	for (int f = 0; f != NF; ++f) {
		dU[f][idx] = 0.0;
		U0[f][idx] = U[f][idx];
	}

}

__global__
void cuda_flux(real* U_base, du_real* dU_base, real dx, int dim, int di,
		real* avisc) {

	int nx, ny, nz;
	int xi, yi, zi;
	int nx1, ny1, nz1;

	__shared__
	extern real shared_real[];

	switch (dim) {
	case XDIM:
		nx = blockDim.x;
		ny = gridDim.x;
		nz = gridDim.y;
		xi = threadIdx.x;
		yi = blockIdx.x;
		zi = blockIdx.y;
		nx1 = nx + 2 * BW - 1;
		ny1 = ny + 2 * BW;
		nz1 = nz + 2 * BW;
		break;
	case YDIM:
		ny = blockDim.x;
		nx = gridDim.x;
		nz = gridDim.y;
		yi = threadIdx.x;
		xi = blockIdx.x;
		zi = blockIdx.y;
		nx1 = nx + 2 * BW;
		ny1 = ny + 2 * BW - 1;
		nz1 = nz + 2 * BW;
		break;
		/*case ZDIM:*/
	default:
		nz = blockDim.x;
		nx = gridDim.x;
		ny = gridDim.y;
		zi = threadIdx.x;
		xi = blockIdx.x;
		yi = blockIdx.y;
		nx1 = nx + 2 * BW;
		ny1 = ny + 2 * BW;
		nz1 = nz + 2 * BW - 1;
	}

	const int idx = (xi + BW) + nx1 * (yi + BW) + (nx1 * ny1) * (zi + BW);
	const int sz = (nx1) * (ny1) * (nz1);
	real F[NF];
	real * U[NF];
	du_real* dU[NF];
	for (int f = 0; f != NF; ++f) {
		U[f] = U_base + f * sz;
		dU[f] = dU_base + f * sz;
	}
	const int mom_dim = mom_i + dim;
	if (threadIdx.x == 0) {
		for (int f = 0; f != NF; ++f) {
			U[f][idx - 2 * di] = U[f][idx - di] = U[f][idx];
		}
		U[mom_dim][idx - 2 * di] = U[mom_dim][idx - di] = fmax(0.0,
				U[mom_dim][idx]);
	} else if (threadIdx.x == blockDim.x - 2) {
		for (int f = 0; f != NF; ++f) {
			U[f][idx + 2 * di] = U[f][idx + di] = U[f][idx];
		}
		U[mom_dim][idx + 2 * di] = U[mom_dim][idx + di] = fmin(0.0,
				U[mom_dim][idx]);
	}
	__syncthreads();

	real dxinv = real(1.0) / dx;
	real slm, slp;
	real vm2[NF], vm1[NF], vp1[NF], vp2[NF];
	real UR[NF], UL[NF];
	real ar, al;
	real pl, pr;
	real vl, vr;
	real cl, cr;
	real ekr, ekl;
	real eir, eil;
	real a;
	const int im2 = idx - 2 * di;
	const int im1 = idx - di;
	const int ip1 = idx;
	const int ip2 = idx + di;
	vm2[den_i] = U[den_i][im2];
	vm1[den_i] = U[den_i][im1];
	vp1[den_i] = U[den_i][ip1];
	vp2[den_i] = U[den_i][ip2];
	const real rho_m2_inv = 1.0 / U[den_i][im2];
	const real rho_m1_inv = 1.0 / U[den_i][im1];
	const real rho_p1_inv = 1.0 / U[den_i][ip1];
	const real rho_p2_inv = 1.0 / U[den_i][ip2];
	for (int f = 1; f < NF; ++f) {
		vm2[f] = U[f][im2] * rho_m2_inv;
		vm1[f] = U[f][im1] * rho_m1_inv;
		vp1[f] = U[f][ip1] * rho_p1_inv;
		vp2[f] = U[f][ip2] * rho_p2_inv;
	}
	for (int f = 0; f != NF; ++f) {
		slm = minmod_theta(vm1[f] - vm2[f], vp1[f] - vm1[f]);
		slp = minmod_theta(vp1[f] - vm1[f], vp2[f] - vp1[f]);
		UL[f] = vm1[f] + 0.5 * slm;
		UR[f] = vp1[f] - 0.5 * slp;
	}
	for (int f = 1; f < NF; ++f) {
		UL[f] *= UL[den_i];
		UR[f] *= UR[den_i];
	}
	ekr = ekl = 0.0;
	for (int d = 0; d != NDIM; ++d) {
		ekl += UL[mom_i + d] * UL[mom_i + d];
		ekr += UR[mom_i + d] * UR[mom_i + d];
	}
	const real rhoLinv = real(1.0) / UL[den_i];
	const real rhoRinv = real(1.0) / UR[den_i];
	ekl *= 0.5 * rhoLinv;
	ekr *= 0.5 * rhoRinv;
	const real etl = UL[ene_i];
	const real etr = UR[ene_i];
	eil = etl - ekl;
	eir = etr - ekr;
	if (eil < etl * DE_SWITCH_1) {
		eil = pow(UL[tau_i], FGAMMA);
	}
	if (eir < etr * DE_SWITCH_1) {
		eir = pow(UR[tau_i], FGAMMA);
	}
	pl = (FGAMMA - 1.0) * eil;
	pr = (FGAMMA - 1.0) * eir;
	cl = sqrt(FGAMMA * pl * rhoLinv);
	cr = sqrt(FGAMMA * pr * rhoRinv);
	vl = UL[mom_dim] * rhoLinv;
	vr = UR[mom_dim] * rhoRinv;
	al = fabs(vl) + cl;
	ar = fabs(vr) + cr;
	a = fmax(al, ar);
	for (int f = 0; f != NF; ++f) {
		F[f] = vr * UR[f] + vl * UL[f] - a * (UR[f] - UL[f]);
	}
	F[mom_dim] += pl + pr;
	F[ene_i] += vl * pl + vr * pr;
	for (int f = 0; f != NF; ++f) {
		F[f] *= 0.5;
	}
	const real x = devXf[XDIM][idx];
	const real y = devXf[YDIM][idx];
	const real z = devXf[ZDIM][idx];
	const real xcp = devXc[XDIM][idx];
	const real ycp = devXc[YDIM][idx];
	const real zcp = devXc[ZDIM][idx];
	const real xcm = devXc[XDIM][idx - di];
	const real ycm = devXc[YDIM][idx - di];
	const real zcm = devXc[ZDIM][idx - di];
	const real h3 = hhh(x, y, z);
	const real hjinv = real(1) / h(dim, x, y, z);
	const real hjinv2 = hjinv * hjinv;
	const real h3pinv = real(1) / hhh(xcp, ycp, zcp);
	const real h3minv = real(1) / hhh(xcm, ycm, zcm);
	const real factorp = dxinv * h3 * hjinv2 * h3pinv;
	const real factorm = dxinv * h3 * hjinv2 * h3minv;
	for (int f = 0; f != NF; ++f) {
		atomicAdd(&(dU[f][idx]), du_real(F[f] * factorp));
		atomicAdd(&(dU[f][idx - di]), -du_real(F[f] * factorm));
	}
	if (avisc != nullptr) {
		int tid = threadIdx.x
				+ blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
		avisc[tid] = a;
	}

}

__global__
void cuda_advance(real* U_base, real* U0_base, du_real* dU_base, real dt,
		real beta) {
	const int nx1 = blockDim.x + 2 * BW;
	const int ny1 = gridDim.x + 2 * BW;
	const int nz1 = gridDim.y + 2 * BW;
	const int xi = threadIdx.x;
	const int yi = blockIdx.x;
	const int zi = blockIdx.y;
	const int idx = (xi + BW) + nx1 * (yi + BW) + (nx1 * ny1) * (zi + BW);
	const int sz = (nx1) * (ny1) * (nz1);
	real * U[NF];
	real * U0[NF];
	du_real* dU[NF];
	for (int f = 0; f != NF; ++f) {
		U[f] = U_base + f * sz;
		U0[f] = U0_base + f * sz;
		dU[f] = dU_base + f * sz;
	}
	for (int f = 0; f != NF; ++f) {
		U[f][idx] += beta * dt * real(dU[f][idx])
				+ (U0[f][idx] - U[f][idx]) * (1.0 - beta);
		dU[f][idx] = du_real(0.0);
	}
	real ek = 0.0;
	const real rhoinv = real(1.0) / U[den_i][idx];
	for (int dim = 0; dim != NDIM; ++dim) {
		ek += U[mom_i + dim][idx] * U[mom_i + dim][idx];
	}
	ek *= 0.5 * rhoinv;
	const real etot = U[ene_i][idx];
	const real ei = etot - ek;
	if (ei > etot * DE_SWITCH_2) {
		U[tau_i][idx] = pow(ei, 1.0 / FGAMMA);
	}
}

__global__
void cuda_source(real* U_base, du_real* dU_base) {
	const int nx1 = blockDim.x + 2 * BW;
	const int ny1 = gridDim.x + 2 * BW;
	const int nz1 = gridDim.y + 2 * BW;
	const int xi = threadIdx.x;
	const int yi = blockIdx.x;
	const int zi = blockIdx.y;
	const int idx = (xi + BW) + nx1 * (yi + BW) + (nx1 * ny1) * (zi + BW);
	const int sz = (nx1) * (ny1) * (nz1);
	real * U[NF];
	du_real* dU[NF];
	for (int f = 0; f != NF; ++f) {
		U[f] = U_base + f * sz;
		dU[f] = dU_base + f * sz;
	}
#ifdef CARTESIAN
#endif
#ifdef CYLINDRICAL
	real ek = 0.0;
	for (int d = 0; d != NDIM; ++d) {
		ek += U[mom_i + d][idx] * U[mom_i + d][idx];
	}
	const real rhoinv = real(1.0) / U[den_i][idx];
	ek *= 0.5 * rhoinv;
	const real et = U[ene_i][idx];
	real ei = et - ek;
	if (ei < et * DE_SWITCH_1) {
		ei = pow(U[tau_i][idx], FGAMMA);
	}
	real p = (FGAMMA - 1.0) * ei;
	const real Rinv = 1.0 / devXc[XDIM][idx];
	const real R3inv = Rinv * Rinv * Rinv;
	dU[mom_i + XDIM][idx] += p * Rinv;
	dU[mom_i + YDIM][idx] += U[den_i][idx] * U[mom_i + YDIM][idx]
			* U[mom_i + YDIM][idx] * R3inv;
#endif
#ifdef SPHERICAL
#endif

}

real cuda_hydro_wrapper(real* rho, real* s[NDIM], real* egas, int nx, int ny,
		int nz, real dx, real dy, real dz) {
	static bool first_call = true;
	const int sz = nx * ny * nz;

	static real* U;
	static real* U0;
	static du_real* dU;
	static real* avisc[NDIM];
	static real* local_avisc[NDIM];
	static real dX[] = { dx, dy, dz };

	static dim3 blocks[NDIM];
	static dim3 threads[NDIM];
	static dim3 blocks0(ny - 2 * BW, nz - 2 * BW);
	static dim3 threads0(nx - 2 * BW);

	real dt;

	if (first_call) {
		create_coordinates(dx, dy, dz, nx, ny, nz);
		blocks[XDIM] = dim3(ny - 2 * BW, nz - 2 * BW);
		blocks[YDIM] = dim3(nx - 2 * BW, nz - 2 * BW);
		blocks[ZDIM] = dim3(nx - 2 * BW, ny - 2 * BW);
		threads[XDIM] = dim3(nx - 2 * BW + 1);
		threads[YDIM] = dim3(ny - 2 * BW + 1);
		threads[ZDIM] = dim3(nz - 2 * BW + 1);
		for (int dim = 0; dim != NDIM; ++dim) {
			const int this_sz = blocks[dim].x * blocks[dim].y * threads[dim].x;
			hipMalloc(&(avisc[dim]), this_sz * sizeof(real));
			local_avisc[dim] = new real[this_sz];
		}
		hipMalloc(&U, NF * sz * sizeof(real));
		hipMalloc(&U0, NF * sz * sizeof(real));
		hipMalloc(&dU, NF * sz * sizeof(du_real));
		first_call = false;
	}

	hipMemcpy(U + sz * den_i, rho, sz * sizeof(real), hipMemcpyHostToDevice);
	hipMemcpy(U + sz * ene_i, egas, sz * sizeof(real), hipMemcpyHostToDevice);
	for (int d = 0; d != NDIM; ++d) {
		hipMemcpy(U + sz * (mom_i + d), s[d], sz * sizeof(real),
				hipMemcpyHostToDevice);
	}

	const int di[NDIM] = { 1, nx, nx * ny };

	INVOKE2(cuda_prep, blocks0, threads0, U, U0, dU);

	std::future < real > dt_fut[NDIM];

	dt = std::numeric_limits < real > ::max();

	for (int rk = 0; rk < NRK; ++rk) {
		const real beta = rk == 0 ? 1.0 : 0.5;

		for (int dim = 0; dim != NDIM; ++dim) {
			dt_fut[dim] =
					std::async(std::launch::async,
							[=]() {
								INVOKE2(cuda_flux, (blocks[dim]),(threads[dim]),U, dU, dX[dim], dim, (di[dim]), (rk == 0 ? avisc[dim] : nullptr));
								real dt = std::numeric_limits<real>::max();
								if (rk == 0) {
									const int this_sz = blocks[dim].x * blocks[dim].y * threads[dim].x;
									hipMemcpy(local_avisc[dim], avisc[dim], this_sz * sizeof(real),
											hipMemcpyDeviceToHost);
									for (int b = 0; b < this_sz; ++b) {
										dt = std::min(dt, CFL * dX[dim] / local_avisc[dim][b]);
									}
								}
								return dt;
							});
		}

		for (int dim = 0; dim != NDIM; ++dim) {
			const real this_dt = dt_fut[dim].get();
			if (rk == 0) {
				dt = std::min(dt, this_dt);
			}
		}
		/**/INVOKE2(cuda_advance, blocks0, threads0, U, U0, dU, dt, beta);

		/**/}

	hipMemcpy(rho, U + sz * den_i, sz * sizeof(real), hipMemcpyDeviceToHost);
	hipMemcpy(egas, U + sz * ene_i, sz * sizeof(real), hipMemcpyDeviceToHost);
	for (int d = 0; d != NDIM; ++d) {
		hipMemcpy(s[d], U + sz * (mom_i + d), sz * sizeof(real),
				hipMemcpyDeviceToHost);
	}

	return dt;
}
